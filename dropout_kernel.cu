#include "hip/hip_runtime.h"
#include "dropout_kernel.h"
#include <hiprand/hiprand_kernel.h>

#define BLOCK_SIZE 16

__device__ int WangHash(int a) {
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);
    return a;
}

__global__ void dropout_kernel (const float * __restrict__ src,
                                float * __restrict__ dst, float * __restrict__ dst_idx, int m, int n, float p){
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    if (row < m && col < n){

        // hiprand_init is very slow.
        // so we use the technique as bellow.
        // http://richiesams.blogspot.jp/2015/03/creating-randomness-and-acummulating.html
        // or https://devtalk.nvidia.com/default/topic/480586/hiprand-initialization-time/
        //generate random number
        int SEED = WangHash(1234);
        hiprandState_t state;
        int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
        //hiprand_init( (SEED << 20) + threadId, 0, 0, &state);
        hiprand_init( SEED + threadId, 0, 0, &state);
        float randNum = hiprand_uniform(&state);


        if (randNum > p){
            dst[row * n + col] = src[row * n + col];
            dst_idx[row * n + col] = 1.0f;
        }
        else{
            dst[row * n + col] = 0.0f;
            dst_idx[row * n + col] = 0.0f;
        }

    }

}

void dropout_kernel_exec(const float *src, float *dst, float *dst_idx, int m, int n, float p){
    /* specified block and grid size */
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((n+block.x-1)/block.x, (m+block.y-1)/block.y);

    /* lunch kernel */
    dropout_kernel<<<grid, block>>>(src, dst, dst_idx, m, n, p);
    hipDeviceSynchronize();
}
